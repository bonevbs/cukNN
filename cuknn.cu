#include "hip/hip_runtime.h"
#include <stdio.h>
#include "npy.hpp"

using namespace std;

//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {
  for(int i = 0; i < nr_rows_A; ++i){
    for(int j = 0; j < nr_cols_A; ++j){
      cout << A[j * nr_rows_A + i] << " ";
    }
    cout << endl;
  }
  cout << endl;
}

__global__ void compute_distance_matrix(vector<vector<double>> & reference_points, vector<vector<double>> & query_points) {

}

void cuda_knn_search(vector<vector<double>> & reference_points, vector<vector<double>> & query_points) {
  
}

vector<vector<double>> read_points(const string filename) {

  bool fortran_order = false;
  vector<unsigned long> shape;
  vector<double> data_buffer;
  vector<vector<double>> data;
  shape.clear();
  data_buffer.clear();

  cout << "Reading point data from " << filename << endl;
  npy::LoadArrayFromNumpy(filename, shape, fortran_order, data_buffer);
  if (shape.size() != 2) {
    throw runtime_error("Dimension mismatch. Expected two-dimensional array.");
  }
  cout << "Point data has dimensions: " << shape[0] << ", " << shape[1] << endl;

  // reshape the array anbd copy it to data
  for (size_t i = 0; i < shape[0]; i++) {
    vector<double> point(shape[1]);
    for (size_t j = 0; j < shape[1]; j++) {
      point[j] = data_buffer[i*shape[1] + j];
    }
    data.push_back(point);
  }

  return data;
}

int main(int argc, char **argv) {

  // parse input file name and read the input file
  if (argc < 3) {
      cout << "Expected two arguments, one for the reference point data and one for the query points." << endl;
      return -1;
  }

  // do the reading
  vector<vector<double>> reference_points = read_points(argv[1]);
  vector<vector<double>> query_points = read_points(argv[2]);

  // prepare datatypes for matrix multiplication
  int rowsR = reference_points.size();
  int colsQ = query_points.size();
  int dims = reference_points[0].size();

  float *h_R = (float *)malloc(rowsR * dims * sizeof(float));
  float *h_Q = (float *)malloc(dims * colsQ * sizeof(float));
  float *h_D = (float *)malloc(rowsR * colsQ * sizeof(float));

  // allocate the arrays on the GPU
  float *d_R, *d_Q, *d_D;
  hipMalloc(&d_R, rowsR * dims * sizeof(float));
  hipMalloc(&d_Q, dims * colsQ * sizeof(float));
  hipMalloc(&d_D, rowsR * colsQ * sizeof(float));

  // free GPU memory
  hipFree(d_R);
  hipFree(d_Q);
  hipFree(d_D);

  // free CPU memory
  free(h_R);
  free(h_Q);
  free(h_D);
  
  return 0;
}